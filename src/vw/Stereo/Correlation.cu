#include "hip/hip_runtime.h"
// __BEGIN_LICENSE__
//  Copyright (c) 2006-2013, United States Government as represented by the
//  Administrator of the National Aeronautics and Space Administration. All
//  rights reserved.
//
//  The NASA Vision Workbench is licensed under the Apache License,
//  Version 2.0 (the "License"); you may not use this file except in
//  compliance with the License. You may obtain a copy of the License at
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
// __END_LICENSE__

#include <vw/Math/BBox.h>
#include <vw/Image/Statistics.h>
#include <vw/Core/Exception.h>
#include <vw/Stereo/Correlation.h>
#include <vw/Stereo/Algorithms.h>
#include <vw/Core/Stopwatch.h>
#include <vw/Image/AlgorithmFunctions.h>
#include <vw/FileIO/DiskImageView.h>

namespace vw { namespace stereo {

  /// Lower level implementation function for calc_disparity.
  /// - The inputs must already be rasterized to safe sizes!
  /// - Since the inputs are rasterized, they must not be too big.
  template <class CostFuncT, class PixelT>
  ImageView<PixelMask<Vector2i>>
  best_of_search_convolution(ImageView<PixelT> const& left_raster,
                             ImageView<PixelT> const& right_raster,
                             BBox2i            const& left_region,
                             Vector2i          const& search_volume,
                             Vector2i          const& kernel_size) {
    
    typedef ImageView<PixelT> ImageType;
    typedef typename CostFuncT::accumulator_type AccumChannelT;
    typedef typename PixelChannelCast<PixelT,AccumChannelT>::type AccumT;
    typedef typename std::pair<AccumT,AccumT> QualT;

    // Build cost function which sometimes has side car data
    CostFuncT cost_function(left_raster, right_raster, kernel_size);

    // Result buffers
    Vector2i result_size = bounding_box(left_raster).size() - kernel_size + Vector2i(1,1);
    ImageView<PixelMask<Vector2i>> disparity_map(result_size[0], result_size[1]);
    std::fill(disparity_map.data(), disparity_map.data() + prod(result_size),
              PixelMask<Vector2i>(Vector2i()));
    // First channel is best, second is worst
    ImageView<QualT> quality_map(result_size[0], result_size[1]);
    
    // Storage buffers
    ImageView<AccumT> cost_metric      (result_size[0], result_size[1]);
    ImageView<AccumT> cost_applied     (left_raster.cols(), left_raster.rows());
    ImageView<PixelT> right_raster_crop(left_raster.cols(), left_raster.rows());

    // Loop across the disparity range we are searching over.
    Vector2i disparity (0, 0);
    for (disparity.y() = 0; disparity.y() != search_volume[1]; ++disparity.y()) {
      for (disparity.x() = 0; disparity.x() != search_volume[0]; ++disparity.x()) {
      
        // Compute correlations quickly by shifting the right image by the
        //  current disparity, computing the pixel difference at each location,
        //  and using fast_box_sum/cost_function to get the final convolution
        //  value at each location in "cost_metric"
      
        // There's only one raster here. Fast box sum calls each pixel
        // individually by pixel accessor. It only calls each pixel
        // once so there's no reason to copy/rasterize the cost result before hand.
        //
        // The cost function should also not be applying an edge
        // extension as we've already over cropped the input.
        
        right_raster_crop = crop(right_raster, bounding_box(left_raster)+disparity);
        cost_applied      = cost_function(left_raster, right_raster_crop);
        cost_metric       = fast_box_sum<AccumChannelT>(cost_applied, kernel_size);
        cost_function.cost_modification(cost_metric, disparity);

        // Loop across the region we want to compute disparities for.
        // - The correlation score for each pixel is located in "cost_metric"
        // - We update the best and worst disparity for each pixel in "quality_map"

        // These conditionals might be served outside of the iteration
        // of dx and dy. It would make the code slightly longer but
        // would avoid a conditional inside a double loop.
        const AccumT* cost_ptr     = cost_metric.data();
        const AccumT* cost_ptr_end = cost_metric.data() + prod(result_size);
        QualT* quality_ptr         = quality_map.data();
        PixelMask<Vector2i>* disparity_ptr = disparity_map.data();
        if (disparity != Vector2i(0,0)) {
          // Normal comparison operations
          while (cost_ptr != cost_ptr_end) {
            if (cost_function.quality_comparison(*cost_ptr, quality_ptr->first)) {
              // Better than best?
              quality_ptr->first = *cost_ptr;
              disparity_ptr->child() = disparity;
            } else if (!cost_function.quality_comparison(*cost_ptr, quality_ptr->second)) {
              // Worse than worse
              quality_ptr->second = *cost_ptr;
            }
            ++cost_ptr;
            ++quality_ptr;
            ++disparity_ptr;
          }
        } else {
          // Initializing quality_map and disparity_map with first result
          while (cost_ptr != cost_ptr_end) {
            quality_ptr->first = quality_ptr->second = *cost_ptr;
            ++cost_ptr;
            ++quality_ptr;
          }
        }
      } // End x loop
    } // End y loop

    // Determine validity of result (detects rare invalid cases)
    size_t invalid_count = 0;
    const QualT* quality_ptr      = quality_map.data();
    const QualT* quality_ptr_end  = quality_map.data() + prod(result_size);
    PixelMask<Vector2i>* disp_ptr = disparity_map.data();
    while (quality_ptr != quality_ptr_end) {
      if (quality_ptr->first == quality_ptr->second) {
        invalidate(*disp_ptr);
        ++invalid_count;
      }
      ++quality_ptr;
      ++disp_ptr;
    }
    //std::cout << "Invalidated " << invalid_count << " pixels in best_of_search_convolution2\n";

    return disparity_map;
  } // End function best_of_search_convolution
 
bool subdivide_regions(ImageView<PixelMask<Vector2i> > const& disparity,
                       BBox2i const& current_bbox,
                       std::vector<SearchParam>& list,
                       Vector2i const& kernel_size,
                       int32 fail_count) {

    // Looking at the 2d disparity vectors inside current_bbox

    const int MIN_REGION_SIZE = 16;

    // 1.) Is this region too small? Must we stop?
    if ( prod(current_bbox.size()) <= 200 ||
         current_bbox.width() < MIN_REGION_SIZE || current_bbox.height() < MIN_REGION_SIZE ){
      BBox2i expanded = current_bbox;
      expanded.expand(1);
      expanded.crop( bounding_box( disparity ) );
      PixelAccumulator<EWMinMaxAccumulator<Vector2i> > accumulator;
      for_each_pixel( crop(disparity, expanded), accumulator );
      if ( !accumulator.is_valid() ) return true;

      list.push_back( SearchParam( current_bbox,
                                   BBox2i(accumulator.minimum(),
                                          accumulator.maximum() + Vector2i(1,1) ) ) );
      return true;
    }

    // 2) Divide the current_bbox into 4 quadrants, does it reduce total search?
    Vector2i split_pt = current_bbox.size()/2;
    BBox2i q1( current_bbox.min(), current_bbox.min()+split_pt );
    BBox2i q4( current_bbox.min()+split_pt, current_bbox.max() );
    BBox2i q2( current_bbox.min() + Vector2i(split_pt[0],0),
               Vector2i(current_bbox.max()[0],current_bbox.min()[1]+split_pt[1]) );
    BBox2i q3( current_bbox.min() + Vector2i(0,split_pt[1]),
               Vector2i(current_bbox.min()[0]+split_pt[0],current_bbox.max()[1]) );
    BBox2i q1_search, q2_search, q3_search, q4_search;

    // Inside each of the four quadrants, find the min and max disparity.
    // - Masked out pixels are ignored
    // - Accumulate product of disparity search region + pixel area
    // - TODO: Should get some of this logic into class functions.
    int32 split_search = 0;
    { // Q1
      PixelAccumulator<EWMinMaxAccumulator<Vector2i> > accumulator;
      for_each_pixel( crop(disparity,q1), accumulator );
      if ( accumulator.is_valid() ) {
        q1_search = BBox2i(accumulator.minimum(),
                           accumulator.maximum()+Vector2i(1,1));
        split_search += q1_search.area() * prod(q1.size()+kernel_size);
      }
    }
    { // Q2
      PixelAccumulator<EWMinMaxAccumulator<Vector2i> > accumulator;
      for_each_pixel( crop(disparity,q2), accumulator );
      if ( accumulator.is_valid() ) {
        q2_search = BBox2i(accumulator.minimum(),
                           accumulator.maximum()+Vector2i(1,1));
        split_search += q2_search.area() * prod(q2.size()+kernel_size);
      }
    }
    { // Q3
      PixelAccumulator<EWMinMaxAccumulator<Vector2i> > accumulator;
      for_each_pixel( crop(disparity,q3), accumulator );
      if ( accumulator.is_valid() ) {
        q3_search = BBox2i(accumulator.minimum(),
                           accumulator.maximum()+Vector2i(1,1));
        split_search += q3_search.area() * prod(q3.size()+kernel_size);
      }
    }
    { // Q4
      PixelAccumulator<EWMinMaxAccumulator<Vector2i> > accumulator;
      for_each_pixel( crop(disparity,q4), accumulator );
      if ( accumulator.is_valid() ) {
        q4_search = BBox2i(accumulator.minimum(),
                           accumulator.maximum()+Vector2i(1,1));
        split_search += q4_search.area() * prod(q4.size()+kernel_size);
      }
    }
    // Now we have an estimate of the cost of processing these four
    // quadrants separately

    // 3) Find current search v2
    //    - Get the min and max disparity search range that we just calculated
    //      for the four quadrants.  This is faster than recomputing the min/max.
    BBox2i current_search_region;
    if ( q1_search != BBox2i() )
      current_search_region = q1_search;
    if ( q2_search != BBox2i() && current_search_region == BBox2i() )
      current_search_region = q2_search;
    else
      current_search_region.grow(q2_search);
    if ( q3_search != BBox2i() && current_search_region == BBox2i() )
      current_search_region = q3_search;
    else
      current_search_region.grow(q3_search);
    if ( q4_search != BBox2i() && current_search_region == BBox2i() )
      current_search_region = q4_search;
    else
      current_search_region.grow(q4_search);
    
    int32 current_search = current_search_region.area() * prod(current_bbox.size()+kernel_size);

    const double IMPROVEMENT_RATIO = 0.8;

    if ( split_search > current_search*IMPROVEMENT_RATIO && fail_count == 0 ) {
      // Splitting up the disparity region did not reduce our workload.
      // This is our first failure, so see if we can still improve by
      //  subdividing the quadrants one more time.
      std::vector<SearchParam> failed;
      if (!subdivide_regions( disparity, q1, list, kernel_size, fail_count + 1 ) )
        failed.push_back(SearchParam(q1,q1_search));
      if (!subdivide_regions( disparity, q2, list, kernel_size, fail_count + 1 ) )
        failed.push_back(SearchParam(q2,q2_search));
      if (!subdivide_regions( disparity, q3, list, kernel_size, fail_count + 1 ) )
        failed.push_back(SearchParam(q3,q3_search));
      if (!subdivide_regions( disparity, q4, list, kernel_size, fail_count + 1 ) )
        failed.push_back(SearchParam(q4,q4_search));
              
      if ( failed.size() == 4 ) {
        // All failed, push back this region as a whole (what we started with)
        list.push_back( SearchParam( current_bbox,
                                     current_search_region ) );
        return true;
      } else if ( failed.size() == 3 ) {
        // 3 failed to split can I merge ?
        // - See the failed==2 case for description!
        std::vector<SearchParam>::const_iterator it1 = failed.begin(), it2 = failed.begin();
        ++it2;
        if ( ( it1->first.min().x() == it2->first.min().x() ||
               it1->first.min().y() == it2->first.min().y() ) &&
             it1->second == it2->second ) {
          BBox2i merge = it1->first;
          merge.grow(it2->first);
          list.push_back( SearchParam( merge, it1->second ) );
          list.push_back( *++it2 );
          return true;
        }
        ++it1; ++it2;
        if ( ( it1->first.min().x() == it2->first.min().x() ||
               it1->first.min().y() == it2->first.min().y() ) &&
             it1->second == it2->second ) {
          BBox2i merge = it1->first;
          merge.grow(it2->first);
          list.push_back( SearchParam( merge, it1->second ) );
          list.push_back( failed.front() );
          return true;
        }
        it1 = failed.begin();
        if ( ( it1->first.min().x() == it2->first.min().x() ||
               it1->first.min().y() == it2->first.min().y() ) &&
             it1->second == it2->second ) {
          BBox2i merge = it1->first;
          merge.grow(it2->first);
          list.push_back( SearchParam( merge, it1->second ) );
          list.push_back( *++it1 );
          return true;
        }
        // Push only the bombed regions, possibly a merge step could go here
        list.insert( list.end(), failed.begin(), failed.end() );
      } else if ( failed.size() == 2 ) {
        // 2 failed to split.
        // If the quadrants are adjacent and have the same disparity range,
        //  merge them into a single search region.
        // - TODO: How often does this actually work?
        if ( ( failed.front().first.min().x() == failed.back().first.min().x() ||
               failed.front().first.min().y() == failed.back().first.min().y() ) &&
             failed.front().second == failed.back().second ) {
          BBox2i merge = failed.front().first;
          merge.grow(failed.back().first);
          list.push_back( SearchParam( merge, failed.front().second ) );
          return true;
        }
        list.insert( list.end(), failed.begin(), failed.end() );
      } else if ( failed.size() == 1 ) {
        // Only 1 failed to split, use it in its entirety, allowing
        // us to take advantage of the other regions which split well.
        list.push_back( failed.front() );
      }
      return true;
    } else if ( split_search > current_search*IMPROVEMENT_RATIO && fail_count > 0 ) {
      // Second failure trying to split this region, give up!
      return false;
    } else {
      // Good split, Try to keep splitting each of the four quadrants further.
      subdivide_regions( disparity, q1, list, kernel_size );
      subdivide_regions( disparity, q2, list, kernel_size );
      subdivide_regions( disparity, q3, list, kernel_size );
      subdivide_regions( disparity, q4, list, kernel_size );
    }
    return true;
  }
  
  ImageView<PixelMask<Vector2i>>
  calc_disparity(CostFunctionType cost_type,
                 ImageViewRef<PixelGray<float>> const& left_in,
                 ImageViewRef<PixelGray<float>> const& right_in,
                 // Valid region in the left image
                 BBox2i                 const& left_region,
                 // Max disparity to search in right image
                 Vector2i               const& search_volume,
                 Vector2i               const& kernel_size) {

    // Sanity check the input:
    VW_DEBUG_ASSERT(kernel_size[0] % 2 == 1 && kernel_size[1] % 2 == 1,
                    ArgumentErr() << "calc_disparity: Kernel input not sized with odd values.");
    VW_DEBUG_ASSERT(kernel_size[0] <= left_region.width() &&
                    kernel_size[1] <= left_region.height(),
                    ArgumentErr() << "calc_disparity: Kernel size too large of active region.");
    VW_DEBUG_ASSERT(search_volume[0] > 0 && search_volume[1] > 0,
                    ArgumentErr() << "calc_disparity: Search volume must be greater than 0.");
    VW_DEBUG_ASSERT(left_region.min().x() >= 0 &&  left_region.min().y() >= 0 &&
                    left_region.max().x() <= left_in.impl().cols() &&
                    left_region.max().y() <= left_in.impl().rows(),
                    ArgumentErr() << "calc_disparity: Region not inside left image.");

    typedef PixelGray<float> pix_type; // to save some typing

    // Rasterize input so that we can do a lot of processing on it.
    BBox2i right_region = left_region;
    right_region.max() += search_volume - Vector2i(1,1);
    ImageView<pix_type> left (crop(left_in.impl(),  left_region));
    ImageView<pix_type> right(crop(right_in.impl(), right_region));

    // Call the lower level function with the appropriate cost function type
    switch (cost_type) {
    case CROSS_CORRELATION:
      return best_of_search_convolution<NCCCost<ImageView<pix_type>>, pix_type>
        (left, right, left_region, search_volume, kernel_size);
    case SQUARED_DIFFERENCE:
      return best_of_search_convolution<SquaredCost<ImageView<pix_type>>, pix_type>
        (left, right, left_region, search_volume, kernel_size);
    default: // case ABSOLUTE_DIFFERENCE:
      return best_of_search_convolution<AbsoluteCost<ImageView<pix_type>>, pix_type>
        (left, right, left_region, search_volume, kernel_size);
    }

    return ImageView<PixelMask<Vector2i>>(); // will not be reached
  } // End function calc_disparity

  /// Create fake left and right images and search volume.  Do a fake
  /// disparity calculation. Divide the run-time of this calculation
  /// by left region size times search box size. This will enable us
  /// to estimate how long disparity calculation takes for given cost
  /// function and kernel size.
  double calc_seconds_per_op(CostFunctionType cost_type, Vector2i const& kernel_size){
     
    double elapsed = -1.0;
    double seconds_per_op = -1.0;

    // We don't know what sizes to use to get a reliable time estimate.
    // So increase the size until the time estimate is a second.
    int lsize = 100;
    while (elapsed < 1.0){

      // Below we add kernel_size to ensure the image exceeds the
      // kernel size, for correlation to perform properly.
      lsize = (int)ceil(lsize*1.2) + std::max(kernel_size[0], kernel_size[1]);

      ImageView<PixelGray<float>> fake_left(lsize, lsize);
      for (int col = 0; col < fake_left.cols(); col++){
        for (int row = 0; row < fake_left.rows(); row++){
          fake_left(col, row) = col%2 + 2*(row%5); // some values
        }
      }

      ImageView<PixelGray<float>> fake_right(4*lsize, 4*lsize);
      for (int col = 0; col < fake_right.cols(); col++){
        for (int row = 0; row < fake_right.rows(); row++){
          fake_right(col, row) = 3*(col%7) + row%3; // some values
        }
      }

      BBox2i search_region(0, 0, lsize/5, lsize/5);
      BBox2i left_region = bounding_box(fake_left);

      Stopwatch watch;
      watch.start();
      ImageView<PixelMask<Vector2i>> disparity =
        calc_disparity(cost_type, fake_left, fake_right,
                       left_region, search_region.size(), kernel_size);
      watch.stop();

      // Note: We add an infinitesimal contribution of disparity, lest
      // the compiler tries to optimize away the above calculation due
      // to its result being unused.
      elapsed = watch.elapsed_seconds() + 1e-40*disparity(0, 0).child().x();
      SearchParam params(left_region, search_region);
      seconds_per_op = elapsed/params.search_volume();
    }

    return seconds_per_op;
  }
  
}} // end namespace vw::stereo
