#include "hip/hip_runtime.h"
#include <vw/Image/ImageView.h>
#include <vw/Image/PixelMask.h>
#include <vw/Image/Interpolation.h>
#include <vw/Core/Exception.h>
#include <vw/Stereo/DisparityMap.h>
#include <vw/Stereo/Correlation.h>
#include <vw/Stereo/CorrEval.h>

// See CorrEval.h for documentation.

namespace vw { namespace stereo {

// Calculate left and right patches. It assumes everything was setup properly
// and all the checks have been done.
void calc_patches(// Inputs
                  BBox2i const& bbox, Vector2i const& kernel_size, bool round_to_int,
                  PixelMask<Vector2f> const& disp,
                  BBox2i const& left_box, BBox2i const& right_box,
                  ImageView<PixelMask<float>> const& left,
                  ImageView<PixelMask<float>> const& right,
                  int col, int row, // patches are around this col and row
                  // Outputs
                  ImageView<PixelMask<float>> & left_patch,
                  ImageView<PixelMask<float>> & right_patch) {

  // An invalid pixel value used for edge extension
  PixelMask<float> nodata_pix(0); nodata_pix.invalidate();
  ValueEdgeExtension<PixelMask<float>> nodata_ext(nodata_pix); 
  
  // Interpolate into the right image. Avoid using an ImageViewRef to
  // avoid a per-pixel virtual function overhead. The 'auto' keyword
  // will use the exact type.
  auto interp_right = interpolate(right, BilinearInterpolation(), nodata_ext);
    
  Vector2i half_kernel = kernel_size/2;
  
  // Iterate over the patch
  for (int c = 0; c < kernel_size[0]; c++) {
    for (int r = 0; r < kernel_size[1]; r++) {
      
      // Left pixel and right pixels in the full images.
      // The left pix is int, but the right pix is not because disp is float.
      // Make it double for added precision.
      Vector2i left_pix(col + bbox.min().x() + c - half_kernel[0],
                        row + bbox.min().y() + r - half_kernel[1]);
      Vector2  right_pix = Vector2(left_pix) + Vector2(disp.child());
                                                       
      // Compensate for the fact that we will access cropped image
      // versions (which may either cut or extend the original
      // images).
      left_pix  -= left_box.min();
      right_pix -= right_box.min();

      // Sanity check for left_pix. We do not check right_pix, as maybe filtering
      // messed it up and it went out of bounds. In that case interpolation
      /// will simply return an invalid value.
      if (!bounding_box(left).contains(left_pix))
        vw_throw(ArgumentErr() << "Out of bounds in the NCC calculation. "
                 << "This is not expected.");
      
      left_patch(c, r)  = left(left_pix[0], left_pix[1]);           // access int pix
      
      if (!round_to_int) 
        right_patch(c, r) = interp_right(right_pix[0], right_pix[1]); // interp float pix
      else
        right_patch(c, r) = right(right_pix[0], right_pix[1]); // do not interpolate
    }
  }
}
  
// Calc NCC. Return -1 on failure (normally NCC is non-negative).  
double calc_ncc(ImageView<PixelMask<float>> const& left_patch,
                ImageView<PixelMask<float>> const& right_patch) {

  if (left_patch.cols() != right_patch.cols() || left_patch.rows() != right_patch.rows()) 
    vw_throw(ArgumentErr() << "The left and right patches have different dimensions.");
  
  double num = 0.0, den1 = 0.0, den2 = 0.0;
  for (int c = 0; c < left_patch.cols(); c++) {
    for (int r = 0; r < left_patch.rows(); r++) {
      if (!is_valid(left_patch(c, r) || !is_valid(right_patch(c, r)))) 
        continue;
      
      double a = left_patch(c, r).child();
      double b = right_patch(c, r).child();
      num  += a*b;
      den1 += a*a;
      den2 += b*b;
    }
  }
  
  if (den1 > 0.0 && den2 > 0.0) 
    return  num / sqrt(den1 * den2);
    
  return -1.0;
}

// Calc stddev. Skip invalid pixels. Return -1 on failure (normally
// stddev is non-negative).
double calc_stddev(ImageView<PixelMask<float>> const& patch) {

  // Find the mean
  int num = 0;
  double mean = 0.0;
  for (int c = 0; c < patch.cols(); c++) {
    for (int r = 0; r < patch.rows(); r++) {
      if (!is_valid(patch(c, r))) 
        continue;
      
      num  += 1;
      mean += patch(c, r).child();
    }
  }
  
  if (num == 0) 
    return -1.0;

  mean /= num;

  double sum = 0.0;
  num = 0.0;
  
  for (int c = 0; c < patch.cols(); c++) {
    for (int r = 0; r < patch.rows(); r++) {
      if (!is_valid(patch(c, r))) 
        continue;
      
      num += 1;
      sum += (patch(c, r).child() - mean) * (patch(c, r).child() - mean);
    }
  }
  
  if (num == 0) 
    return -1.0;

  return sqrt(sum / num);
}

CorrEval::prerasterize_type CorrEval::prerasterize(vw::BBox2i const& bbox) const {
  
  // Bring the disparity for the given processing region in memory.
  // It was checked before that it has the correct extent.
  ImageView<PixelMask<Vector2f>> disp = crop(m_disp, bbox);

  if (m_round_to_int) {
    for (int col = 0; col < disp.cols(); col++) {
      for (int row = 0; row < disp.rows(); row++) {
        // Round the disparity (both valid and invalid values, the validity is not affected)
        disp(col, row).child() = round(disp(col, row).child());
      }
    }
    
    // TODO(oalexan1): Consider subdividing regions as done in stereo
    // correlation.  For disparities that have integer values, and
    // which vary little over a large area, that may be more efficient
    // than the per-pixel approach. However, if the range of
    // disparities in a region is more than the kernel size, the
    // per-pixel approach should do better. So there has to be a check
    // and much testing.
    
    // Also, the best_of_search_convolution() logic in Correlation.cc
    // needs some modifications, since there the disparity with lowest
    // cost function is kept, but here we must keep the cost function
    // for the given known disparity regardless of cost function
    // value.
    
    //  ImageView<PixelMask<Vector2i>> int_disp(disp.cols(), disp.rows());
    //  std::vector<stereo::SearchParam> zones; 
    //  subdivide_regions(int_disp, bounding_box(int_disp),
    //   zones, m_kernel_size);
    // Now must iterate over regions.
  }

  Vector2i half_kernel = m_kernel_size/2;
  
  // Need to be able to look beyond the current tile in left image
  // to be able to compute the NCC.
  BBox2i left_box = bbox;
  left_box.expand(half_kernel);

  // For the right image it is more complicated. Need to also
  // consider the disparity and interpolation.
  // Note: The memory usage can be high for a large disparity.
  
  // TODO(oalexan1): When finding the curvature of NCC will need to further
  // expand the box given the neighborhood we will use then. 
  BBox2i right_box;
  for (int col = 0; col < disp.cols(); col++) {
    for (int row = 0; row < disp.rows(); row++) {
      if (!is_valid(disp(col, row))) 
        continue;

      if (col % m_sample_rate != 0 || row % m_sample_rate != 0) 
        continue;
      
      Vector2 left_pix  = bbox.min() + Vector2(col, row);
      Vector2 right_pix = left_pix + disp(col, row).child();
      right_box.grow(Vector2(floor(right_pix.x()), floor(right_pix.y())));
      right_box.grow(Vector2(ceil(right_pix.x()), ceil(right_pix.y())));
    }
  }
  
  right_box.expand(half_kernel); // Take into account the kernel
  right_box.expand(BilinearInterpolation::pixel_buffer); // Due to interpolation
  right_box.expand(2); // because right_box is exclusive in the upper-right, and +1 just in case

  // An invalid pixel value used for edge extension
  PixelMask<float> nodata_pix(0); nodata_pix.invalidate();
  ValueEdgeExtension<PixelMask<float>> nodata_ext(nodata_pix); 
  
  // Crop portions of the inputs and bring them in memory. Extend them if need be
  // with invalid data to not go out of range later. Data validity will be checked.
  ImageView<PixelMask<float>> left  = crop(edge_extend(m_left, nodata_ext), left_box);
  ImageView<PixelMask<float>> right = crop(edge_extend(m_right, nodata_ext), right_box);
  
  // Allocate room for the patches
  ImageView<PixelMask<float>> left_patch(m_kernel_size[0], m_kernel_size[1]);
  ImageView<PixelMask<float>> right_patch(m_kernel_size[0], m_kernel_size[1]);

  // Create the tile with the result
  ImageView<result_type> tile(bbox.width(), bbox.height());
  for (int col = 0; col < tile.cols(); col++) {
    for (int row = 0; row < tile.rows(); row++) {
      
      // Start the tile as invalid
      tile(col, row) = PixelMask<float>(0.0);
      tile(col, row).invalidate();
      
      PixelMask<Vector2f> d = disp(col, row);
      if (!is_valid(d))
        continue;

      if (col % m_sample_rate != 0 || row % m_sample_rate != 0) 
        continue;
      
      calc_patches(// Inputs
                   bbox, m_kernel_size, m_round_to_int, d,
                   left_box, right_box,  
                   left, right,  
                   col, row,  // patches are around this col and row
                   // Outputs
                   left_patch, right_patch);
      
      if (m_metric == "ncc") {
        double ncc = calc_ncc(left_patch, right_patch);
        if (ncc >= 0) {
          tile(col, row).validate();
          tile(col, row).child() = ncc;
        }
      } else if (m_metric == "stddev") {
        double left_stddev = calc_stddev(left_patch);
        double right_stddev = calc_stddev(right_patch);
        if (left_stddev >= 0.0 && right_stddev >= 0.0) {
          tile(col, row).validate();
          tile(col, row).child() = (left_stddev + right_stddev)/2.0;
        }
      }
    }
  }

  return prerasterize_type(tile, -bbox.min().x(), -bbox.min().y(),
                           cols(), rows());
}
  
}} // end namespace vw::stereo

